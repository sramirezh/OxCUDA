#include "hip/hip_runtime.h"

////////////////////////////////////////////////////////////////////////
// GPU version of Monte Carlo algorithm using NVIDIA's CURAND library
////////////////////////////////////////////////////////////////////////

#include <stdlib.h>
#include <stdio.h>
#include <math.h>

#include <hip/hip_runtime.h>
#include <hiprand.h>

#include <hip/hip_runtime_api.h>

////////////////////////////////////////////////////////////////////////
// CUDA global constants
////////////////////////////////////////////////////////////////////////

__constant__ int   N;
__constant__ float a,b,c;


////////////////////////////////////////////////////////////////////////
// kernel routinGe
////////////////////////////////////////////////////////////////////////


__global__ void function(float *d_z,float *d_value)
{
  int   ind;

  // move array pointers to correct position

  ind = threadIdx.x +blockIdx.x*blockDim.x;

  // function calculation
	
  d_value[ind]=a*d_z[ind]*d_z[ind]+b*d_z[ind]+c;


//   printf(" ind, d_z, d_value =  %d %f %f \n",ind,d_z[ind],d_value[ind]); 
}


////////////////////////////////////////////////////////////////////////
// Main program
////////////////////////////////////////////////////////////////////////

int main(int argc, const char **argv){
    
  int Npoints=640000;	
  float h_a,h_b,h_c; 
  float   *h_value, *d_value, *d_z;
  double  sum1;

  // initialise card

  findCudaDevice(argc, argv);

  // initialise CUDA timing

  float milli;
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  // allocate memory on host and device

  h_value = (float *)malloc(sizeof(float)*Npoints);

  checkCudaErrors( hipMalloc((void **)&d_value, sizeof(float)*Npoints) );
  checkCudaErrors( hipMalloc((void **)&d_z, sizeof(float)*Npoints) );

  // define constants and transfer to GPU

  h_a     = 1.0f;
  h_b     = 2.0f;
  h_c     = 3.0f;

  checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL(a),    &h_a,    sizeof(h_a)) );
  checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL(b),    &h_b,    sizeof(h_b)) );
  checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL(c),    &h_c,    sizeof(h_c)) );

  // random number generation

  hipEventRecord(start);

  hiprandGenerator_t gen;
  checkCudaErrors( hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT) );
  checkCudaErrors( hiprandSetPseudoRandomGeneratorSeed(gen, 1234ULL) );
  checkCudaErrors( hiprandGenerateNormal(gen, d_z,Npoints, 0.0f, 1.0f) );
 
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&milli, start, stop);

  printf("CURAND normal RNG  execution time (ms): %f,  samples/sec: %e \n",
          milli, Npoints/(0.001*milli));

  // execute kernel and time it

  hipEventRecord(start);

  function<<<Npoints/64, 64>>>(d_z, d_value);
  getLastCudaError("function execution failed\n");

  hipEventRecord(stop);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&milli, start, stop);

  printf("Averaging execution time (ms): %f \n",milli);

  // copy back results

  checkCudaErrors( hipMemcpy(h_value, d_value, sizeof(float)*Npoints,
                   hipMemcpyDeviceToHost) );

  // compute average

  sum1 = 0.0;
  for (int i=0; i<Npoints; i++) {
    sum1 += h_value[i];
  }

  printf("\nAverage value   = %13.8f\n\n",sum1/Npoints);

  // Tidy up library

  checkCudaErrors( hiprandDestroyGenerator(gen) );

  // Release memory and exit cleanly

  free(h_value);
  checkCudaErrors( hipFree(d_value) );
  checkCudaErrors( hipFree(d_z) );

  // CUDA exit -- needed to flush printf write buffer

  hipDeviceReset();

}
